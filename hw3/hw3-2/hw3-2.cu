#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/stat.h> 
#include <sys/types.h>


//======================
#define min__(a, b) ((a) < (b) ? (a) : (b))
#define B 64//block factor of blocked-Floyd Warshall
#define BLOCK_SIZE 32//GPU block size
#define DEV_NO 0
#define INF 0x3FFFFFFF
//cudaDeviceProp prop;
int vertex, edge, V;
int *Dist = NULL;
__device__ int min_(int a, int b) {return min(a, b);} 
int ceil(int a, int b) { return (a + b - 1) / b; }
__global__ void Phase1(int *dst, int Round, int V){
    int i = threadIdx.y;
	int j = threadIdx.x;
	int k=0;
    int i_B = i + BLOCK_SIZE;
	int j_B = j + BLOCK_SIZE;
    //B為64，一次處理一個64*64，但block dim僅為32*32，因此一個大block分成四個小block，一個thread要計算四個小block四個點
    // 1 2
    // 3 4
	int offset = Round * B * (V+1);
    int blk_pt1 = offset + i * V + j;
    int blk_pt2 = offset + i * V + j_B;
    int blk_pt3 = offset + i_B * V + j;
    int blk_pt4 = offset + i_B * V + j_B;

	__shared__ int share[B][B];
    share[i][j] = dst[blk_pt1];
	share[i][j_B] = dst[blk_pt2];
	share[i_B][j] = dst[blk_pt3];
	share[i_B][j_B] = dst[blk_pt4];
	__syncthreads();
	
	while(k<B){
		share[i][j] = min_(share[i][j], share[i][k] + share[k][j]);
		share[i_B][j] = min_(share[i_B][j], share[i_B][k] + share[k][j]);
		share[i][j_B] = min_(share[i][j_B], share[i][k] + share[k][j_B]);
		share[i_B][j_B] = min_(share[i_B][j_B], share[i_B][k] + share[k][j_B]);
        //if(share[i*B+j]>share[i*B+k] + share[k*B+j]) { share[i*B+j]=share[i*B+k] + share[k*B+j];}
        //if(share[i*B+j_B]>share[i*B+k] + share[k*B+j_B]) { share[i*B+j_B]=share[i*B+k] + share[k*B+j_B];}
        //if(share[i_B*B+j]>share[i_B*B+k] + share[k*B+j]) { share[i_B*B+j]=share[i_B*B+k] + share[k*B+j];}
        //if(share[i_B*B+j_B]>share[i_B*B+k] + share[k*B+j_B]) { share[i_B*B+j_B]=share[i_B*B+k] + share[k*B+j_B];}
		++k;
		__syncthreads();
	}
	dst[blk_pt1] = share[i][j];
	dst[blk_pt2] = share[i][j_B];
	dst[blk_pt3] = share[i_B][j];
	dst[blk_pt4] = share[i_B][j_B];
}


__global__ void Phase2_(int *dst, int Round, int V) {
	if (blockIdx.x == Round) {return;}
	
	int i = threadIdx.y;
	int j = threadIdx.x;
	int k=0;
	int offset = Round * B * (V+1);
	int offset_rc, pivot_rc_blk1, pivot_rc_blk2, pivot_rc_blk3, pivot_rc_blk4;
	
	int i_B = i + BLOCK_SIZE;
	int j_B = j + BLOCK_SIZE;
	int blk_pt1 = offset + i * V+ j;
	int blk_pt2 = offset + i * V + j_B;
	int blk_pt3 = offset + i_B * V + j;
	int blk_pt4 = offset + i_B *V + j_B;
	//for same col block
	if(blockIdx.y == 0) {
		offset_rc = blockIdx.x * B * V + Round * B;
		pivot_rc_blk1 = offset_rc + i * V + j;
		pivot_rc_blk2 = offset_rc + i * V + j_B;
		pivot_rc_blk3 = offset_rc + i_B * V + j;
		pivot_rc_blk4 = offset_rc + i_B * V + j_B;
	}else {//for same row block
		offset_rc = Round * B * V + blockIdx.x * B;
		pivot_rc_blk1 = offset_rc + i * V + j;
		pivot_rc_blk2 = offset_rc + i * V + j_B;
		pivot_rc_blk3 = offset_rc + i_B * V + j;
		pivot_rc_blk4 = offset_rc + i_B * V + j_B;
	}

	__shared__ int s[B][B];
	__shared__ int rc_share[B][B];

	s[i][j] = dst[blk_pt1];
	s[i][j_B] = dst[blk_pt2];
	s[i_B][j] = dst[blk_pt3];
	s[i_B][j_B] = dst[blk_pt4];

	rc_share[i][j] = dst[pivot_rc_blk1];
	rc_share[i][j_B] = dst[pivot_rc_blk2];
	rc_share[i_B][j] = dst[pivot_rc_blk3];
	rc_share[i_B][j_B] = dst[pivot_rc_blk4];
	__syncthreads();
	
	if(blockIdx.y == 0){
		while(k<B){
			rc_share[i][j] = min_(rc_share[i][j], rc_share[i][k] + s[k][j]);
			rc_share[i][j_B] = min_(rc_share[i][j_B], rc_share[i][k] + s[k][j_B]);
			rc_share[i_B][j] = min_(rc_share[i_B][j], rc_share[i_B][k] + s[k][j]);
			rc_share[i_B][j_B] = min_(rc_share[i_B][j_B], rc_share[i_B][k] + s[k][j_B]);
			k++;
			__syncthreads();
		}
	} else {
		while(k<B){
			rc_share[i][j] = min_(rc_share[i][j], s[i][k] + rc_share[k][j]);
			rc_share[i][j_B] = min_(rc_share[i][j_B], s[i][k] + rc_share[k][j_B]);
			rc_share[i_B][j] = min_(rc_share[i_B][j], s[i_B][k] + rc_share[k][j]);
			rc_share[i_B][j_B] = min_(rc_share[i_B][j_B], s[i_B][k] + rc_share[k][j_B]);
			k++;
			__syncthreads();
		}
	}

	dst[pivot_rc_blk1] = rc_share[i][j];
	dst[pivot_rc_blk2] = rc_share[i][j_B];
	dst[pivot_rc_blk3] = rc_share[i_B][j];
	dst[pivot_rc_blk4] = rc_share[i_B][j_B];

}

__global__ void Phase3(int *dst, int Round, int V) {
	if (blockIdx.x == Round || blockIdx.y == Round) {return;}
	int i = threadIdx.y;
	int j = threadIdx.x;
	int k=0;
	int i_B = i + BLOCK_SIZE;
	int j_B = j + BLOCK_SIZE;
	
	int offset_ = blockIdx.y * B * V + blockIdx.x * B;
	int blk_pt1 = offset_ + i * V + j;
	int blk_pt2 = offset_ + i * V + j_B;
	int blk_pt3 = offset_ + i_B * V + j;
	int blk_pt4 = offset_ + i_B * V + j_B;
	//same row
	int offset_r = blockIdx.y * B * V + Round * B;
	int row_blk1 = offset_r + i * V + j;
	int row_blk2 = offset_r + i * V + j_B;
	int row_blk3 = offset_r + i_B * V + j;
	int row_blk4 = offset_r + i_B * V + j_B;
	//same col
	int offset_c = Round * B * V + blockIdx.x * B;
	int col_blk1 = offset_c + i * V + j;
	int col_blk2 = offset_c + i * V+ j_B;
	int col_blk3 = offset_c + i_B * V+ j;
	int col_blk4 = offset_c + i_B * V + j_B;

	

	__shared__ int sh[B][B];
	__shared__ int row_s[B][B];
	__shared__ int col_s[B][B];

	sh[i][j] = dst[blk_pt1];
	sh[i][j_B] = dst[blk_pt2];
	sh[i_B][j] = dst[blk_pt3];
	sh[i_B][j_B] = dst[blk_pt4];

	row_s[i][j] = dst[row_blk1];
	row_s[i][j_B] = dst[row_blk2];
	row_s[i_B][j] = dst[row_blk3];
	row_s[i_B][j_B] = dst[row_blk4];

	col_s[i][j] = dst[col_blk1];
	col_s[i][j_B] = dst[col_blk2];
	col_s[i_B][j] = dst[col_blk3];
	col_s[i_B][j_B] = dst[col_blk4];

	__syncthreads();

	
	
	while(k<B){
		sh[i][j] = min_(row_s[i][k] + col_s[k][j], sh[i][j]);
		sh[i][j_B] = min_(row_s[i][k] + col_s[k][j_B], sh[i][j_B]);
		sh[i_B][j] = min_(row_s[i_B][k] + col_s[k][j], sh[i_B][j]);
		sh[i_B][j_B] = min_(row_s[i_B][k] + col_s[k][j_B], sh[i_B][j_B]);
		
		//if(sh[i*B+j]>(row_s[i*B+k] + col_s[k*B+j])) {sh[i*B+j]=row_s[i*B+k] + col_s[k*B+j];}
		//if(sh[i*B+j_B]>(row_s[i*B+k] + col_s[k*B+j_B])) {sh[i*B+j_B]=row_s[i*B+k] + col_s[k*B+j_B];}
		//if(sh[i_B*B+j]>(row_s[i_B*B+k] + col_s[k*B+j])) {sh[i_B*B+j]=row_s[i_B*B+k] + col_s[k*B+j];}
		//if(sh[i_B*B+j_B]>(row_s[i_B*B+k] + col_s[k*B+j_B])) {sh[i_B*B+j_B]=row_s[i_B*B+k] + col_s[k*B+j_B];}
		++k;
		//__syncthreads();
	}
	dst[blk_pt1] = sh[i][j];
	dst[blk_pt2] = sh[i][j_B];
	dst[blk_pt3] = sh[i_B][j];
	dst[blk_pt4] = sh[i_B][j_B];
}


void block_FW(int V) {
	int round = ceil(vertex, B);
    int *dst = NULL;
	int *dst_ = NULL;

    //partition matrix into ceil(V/B) * ceil(V/B) blocks
    int blocks = ceil(V, B);
	dim3 block_dim(BLOCK_SIZE , BLOCK_SIZE);//BLOCK_SIZE=32
	dim3 grid_dim(blocks, blocks);
	dim3 grid_dim2(blocks, 2);
	size_t size = V*V*sizeof(int);
	size_t size_vertex = vertex*vertex*sizeof(int);
	hipHostRegister(Dist, size, hipHostRegisterDefault);
	hipMalloc(&dst, size);
	hipMemcpy(dst, Dist, size, hipMemcpyHostToDevice);
    
	for (int r = 0; r < round; ++r) {
		// phase 1
		Phase1<<<1, block_dim>>>(dst, r, V);
		// phase 2
		//Phase2<<<blocks, block_dim>>>(dst, r, V);
		Phase2_<<<grid_dim2, block_dim>>>(dst, r, V);
		// phase 3
		Phase3<<<grid_dim, block_dim>>>(dst, r, V);
		//Phase3_<<<grid_dim, block_dim>>>(dst, r, V);
	}

	hipMalloc(&dst_, vertex*vertex*sizeof(int));
	for(int i = 0; i < vertex; ++i) {
		hipMemcpy(dst_ + i*vertex, dst + i*V, sizeof(int)*vertex, hipMemcpyDeviceToDevice);
    }

	hipMemcpy(Dist, dst_, size_vertex, hipMemcpyDeviceToHost);
	hipFree(dst);
	hipFree(dst_);
}
inline void input_(char* infile) {
	int file = open(infile, O_RDONLY);
	int *fpt = (int*)mmap(NULL, 2*sizeof(int), PROT_READ, MAP_PRIVATE, file, 0);
  	vertex = fpt[0];
	edge = fpt[1];
	int *pair = (int*)(mmap(NULL, (3 * edge + 2) * sizeof(int), PROT_READ, MAP_PRIVATE, file, 0));

	if (vertex % B){
        V = vertex + (B - vertex % B);//size of total matrix could be filled with B*B blocks
    } else {V = vertex;}
	Dist = (int*)malloc(V*V*sizeof(int));
	

	for (int i = 0; i < V; ++i) {
    	for (int j = 0; j < V; ++j) {
			Dist[i*V+j] = INF;
			if (i == j) Dist[i*V+j] = 0;
		}
    }

	for (int i = 0; i < edge; ++i) {
		Dist[pair[i*3+2]*V+pair[i*3+3]]= pair[i*3+4];//0,1 is vertex and edge
	}
	munmap(pair, (3 * edge + 2) * sizeof(int));//end the mapping
	close(file);
}
inline void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&vertex, sizeof(int), 1, file);
    fread(&edge, sizeof(int), 1, file);
    if (vertex % B){
        V = vertex + (B - vertex % B);//size of total matrix could be filled with B*B blocks
    } else {V = vertex;}
    Dist = (int*)malloc(V*V*sizeof(int));
    //fprintf(stderr, "edge:%d vertex:%d V:%d\n",edge,vertex,V);
    for (int i = 0; i < V; ++i) {
        for (int j = 0; j < V; ++j) {
                if (i == j){Dist[i*V+j] = 0;}
                else {Dist[i*V+j] = INF;}
                 
            }
    }
    

    int pair[3];
    for (int i = 0; i < edge; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*V+pair[1]] = pair[2];
        //printf("%d ",Dist[pair[0]*V+pair[1]]);
    }
    fclose(file);
}

inline void output_(char* outFileName) {
	//int *Dist_ = (int *)malloc(vertex * vertex * sizeof(int));

    FILE* outfile = fopen(outFileName, "w");
	
	fwrite(Dist, sizeof(int), vertex*vertex, outfile);
    fclose(outfile); 
}






int main(int argc, char* argv[]) {
    input_(argv[1]);
    //cudaGetDeviceProperties(&prop, DEV_NO);
    //printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    block_FW(V);
    output_(argv[2]);
    return 0;
}


/*
#include <stdio.h>
#include <stdlib.h>
#include <cuda.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/stat.h> 
#include <sys/types.h>


//======================
#define min__(a, b) ((a) < (b) ? (a) : (b))
#define B 64//block factor of blocked-Floyd Warshall
#define BLOCK_SIZE 32//GPU block size
#define DEV_NO 0
#define INF 0x3FFFFFFF
//cudaDeviceProp prop;
int vertex, edge, V;
int *Dist = NULL;
__device__ int min_(int a, int b) {return min(a, b);} 
int ceil(int a, int b) { return (a + b - 1) / b; }
__global__ void Phase1(int *dst, int Round, int V){
    int i = threadIdx.y;
	int j = threadIdx.x;
	int k=0;
    int i_B = i + BLOCK_SIZE;
	int j_B = j + BLOCK_SIZE;
    //B為64，一次處理一個64*64，但block dim僅為32*32，因此一個大block分成四個小block，一個thread要計算四個小block四個點
    // 1 2
    // 3 4
	int offset = Round * B * (V+1);
    int blk_pt1 = offset + i * V + j;
    int blk_pt2 = offset + i * V + j_B;
    int blk_pt3 = offset + i_B * V + j;
    int blk_pt4 = offset + i_B * V + j_B;

	__shared__ int share[B][B];
    share[i][j] = dst[blk_pt1];
	share[i][j_B] = dst[blk_pt2];
	share[i_B][j] = dst[blk_pt3];
	share[i_B][j_B] = dst[blk_pt4];
	__syncthreads();
	
	while(k<B){
		share[i][j] = min_(share[i][j], share[i][k] + share[k][j]);
		share[i_B][j] = min_(share[i_B][j], share[i_B][k] + share[k][j]);
		share[i][j_B] = min_(share[i][j_B], share[i][k] + share[k][j_B]);
		share[i_B][j_B] = min_(share[i_B][j_B], share[i_B][k] + share[k][j_B]);
        //if(share[i*B+j]>share[i*B+k] + share[k*B+j]) { share[i*B+j]=share[i*B+k] + share[k*B+j];}
        //if(share[i*B+j_B]>share[i*B+k] + share[k*B+j_B]) { share[i*B+j_B]=share[i*B+k] + share[k*B+j_B];}
        //if(share[i_B*B+j]>share[i_B*B+k] + share[k*B+j]) { share[i_B*B+j]=share[i_B*B+k] + share[k*B+j];}
        //if(share[i_B*B+j_B]>share[i_B*B+k] + share[k*B+j_B]) { share[i_B*B+j_B]=share[i_B*B+k] + share[k*B+j_B];}
		++k;
		__syncthreads();
	}
	dst[blk_pt1] = share[i][j];
	dst[blk_pt2] = share[i][j_B];
	dst[blk_pt3] = share[i_B][j];
	dst[blk_pt4] = share[i_B][j_B];
}


__global__ void Phase2_(int *dst, int Round, int V) {
	if (blockIdx.x == Round) {return;}
	
	int i = threadIdx.y;
	int j = threadIdx.x;
	int k=0;
	int offset = Round * B * (V+1);
	int offset_rc, pivot_rc_blk1, pivot_rc_blk2, pivot_rc_blk3, pivot_rc_blk4;
	
	int i_B = i + BLOCK_SIZE;
	int j_B = j + BLOCK_SIZE;
	int blk_pt1 = offset + i * V+ j;
	int blk_pt2 = offset + i * V + j_B;
	int blk_pt3 = offset + i_B * V + j;
	int blk_pt4 = offset + i_B *V + j_B;
	//for same col block
	if(blockIdx.y == 0) {
		offset_rc = blockIdx.x * B * V + Round * B;
		pivot_rc_blk1 = offset_rc + i * V + j;
		pivot_rc_blk2 = offset_rc + i * V + j_B;
		pivot_rc_blk3 = offset_rc + i_B * V + j;
		pivot_rc_blk4 = offset_rc + i_B * V + j_B;
	}else {//for same row block
		offset_rc = Round * B * V + blockIdx.x * B;
		pivot_rc_blk1 = offset_rc + i * V + j;
		pivot_rc_blk2 = offset_rc + i * V + j_B;
		pivot_rc_blk3 = offset_rc + i_B * V + j;
		pivot_rc_blk4 = offset_rc + i_B * V + j_B;
	}

	__shared__ int s[B][B];
	__shared__ int rc_share[B][B];

	s[i][j] = dst[blk_pt1];
	s[i][j_B] = dst[blk_pt2];
	s[i_B][j] = dst[blk_pt3];
	s[i_B][j_B] = dst[blk_pt4];

	rc_share[i][j] = dst[pivot_rc_blk1];
	rc_share[i][j_B] = dst[pivot_rc_blk2];
	rc_share[i_B][j] = dst[pivot_rc_blk3];
	rc_share[i_B][j_B] = dst[pivot_rc_blk4];
	__syncthreads();
	
	if(blockIdx.y == 0){
		while(k<B){
			rc_share[i][j] = min_(rc_share[i][j], rc_share[i][k] + s[k][j]);
			rc_share[i][j_B] = min_(rc_share[i][j_B], rc_share[i][k] + s[k][j_B]);
			rc_share[i_B][j] = min_(rc_share[i_B][j], rc_share[i_B][k] + s[k][j]);
			rc_share[i_B][j_B] = min_(rc_share[i_B][j_B], rc_share[i_B][k] + s[k][j_B]);
			k++;
			__syncthreads();
		}
	} else {
		while(k<B){
			rc_share[i][j] = min_(rc_share[i][j], s[i][k] + rc_share[k][j]);
			rc_share[i][j_B] = min_(rc_share[i][j_B], s[i][k] + rc_share[k][j_B]);
			rc_share[i_B][j] = min_(rc_share[i_B][j], s[i_B][k] + rc_share[k][j]);
			rc_share[i_B][j_B] = min_(rc_share[i_B][j_B], s[i_B][k] + rc_share[k][j_B]);
			k++;
			__syncthreads();
		}
	}

	dst[pivot_rc_blk1] = rc_share[i][j];
	dst[pivot_rc_blk2] = rc_share[i][j_B];
	dst[pivot_rc_blk3] = rc_share[i_B][j];
	dst[pivot_rc_blk4] = rc_share[i_B][j_B];

}

__global__ void Phase3(int *dst, int Round, int V) {
	if (blockIdx.x == Round || blockIdx.y == Round) {return;}
	int i = threadIdx.y;
	int j = threadIdx.x;
	int k=0;
	int i_B = i + BLOCK_SIZE;
	int j_B = j + BLOCK_SIZE;
	
	int offset_ = blockIdx.y * B * V + blockIdx.x * B;
	int blk_pt1 = offset_ + i * V + j;
	int blk_pt2 = offset_ + i * V + j_B;
	int blk_pt3 = offset_ + i_B * V + j;
	int blk_pt4 = offset_ + i_B * V + j_B;
	//same row
	int offset_r = blockIdx.y * B * V + Round * B;
	int row_blk1 = offset_r + i * V + j;
	int row_blk2 = offset_r + i * V + j_B;
	int row_blk3 = offset_r + i_B * V + j;
	int row_blk4 = offset_r + i_B * V + j_B;
	//same col
	int offset_c = Round * B * V + blockIdx.x * B;
	int col_blk1 = offset_c + i * V + j;
	int col_blk2 = offset_c + i * V+ j_B;
	int col_blk3 = offset_c + i_B * V+ j;
	int col_blk4 = offset_c + i_B * V + j_B;

	

	__shared__ int sh[B][B];
	__shared__ int row_s[B][B];
	__shared__ int col_s[B][B];

	sh[i][j] = dst[blk_pt1];
	sh[i][j_B] = dst[blk_pt2];
	sh[i_B][j] = dst[blk_pt3];
	sh[i_B][j_B] = dst[blk_pt4];

	row_s[i][j] = dst[row_blk1];
	row_s[i][j_B] = dst[row_blk2];
	row_s[i_B][j] = dst[row_blk3];
	row_s[i_B][j_B] = dst[row_blk4];

	col_s[i][j] = dst[col_blk1];
	col_s[i][j_B] = dst[col_blk2];
	col_s[i_B][j] = dst[col_blk3];
	col_s[i_B][j_B] = dst[col_blk4];

	__syncthreads();

	
	
	while(k<B){
		sh[i][j] = min_(row_s[i][k] + col_s[k][j], sh[i][j]);
		sh[i][j_B] = min_(row_s[i][k] + col_s[k][j_B], sh[i][j_B]);
		sh[i_B][j] = min_(row_s[i_B][k] + col_s[k][j], sh[i_B][j]);
		sh[i_B][j_B] = min_(row_s[i_B][k] + col_s[k][j_B], sh[i_B][j_B]);
		
		//if(sh[i*B+j]>(row_s[i*B+k] + col_s[k*B+j])) {sh[i*B+j]=row_s[i*B+k] + col_s[k*B+j];}
		//if(sh[i*B+j_B]>(row_s[i*B+k] + col_s[k*B+j_B])) {sh[i*B+j_B]=row_s[i*B+k] + col_s[k*B+j_B];}
		//if(sh[i_B*B+j]>(row_s[i_B*B+k] + col_s[k*B+j])) {sh[i_B*B+j]=row_s[i_B*B+k] + col_s[k*B+j];}
		//if(sh[i_B*B+j_B]>(row_s[i_B*B+k] + col_s[k*B+j_B])) {sh[i_B*B+j_B]=row_s[i_B*B+k] + col_s[k*B+j_B];}
		++k;
		//__syncthreads();
	}
	dst[blk_pt1] = sh[i][j];
	dst[blk_pt2] = sh[i][j_B];
	dst[blk_pt3] = sh[i_B][j];
	dst[blk_pt4] = sh[i_B][j_B];
}


void block_FW(int V) {
	int round = ceil(vertex, B);
    int *dst = NULL;
	int *dst_ = NULL;

    //partition matrix into ceil(V/B) * ceil(V/B) blocks
    int blocks = ceil(V, B);
	dim3 block_dim(BLOCK_SIZE , BLOCK_SIZE);//BLOCK_SIZE=32
	dim3 grid_dim(blocks, blocks);
	dim3 grid_dim2(blocks, 2);
	size_t size = V*V*sizeof(int);
	size_t size_vertex = vertex*vertex*sizeof(int);
	cudaHostRegister(Dist, size, cudaHostRegisterDefault);
	cudaMalloc(&dst, size);
	cudaMemcpy(dst, Dist, size, cudaMemcpyHostToDevice);
    
	for (int r = 0; r < round; ++r) {
		// phase 1
		Phase1<<<1, block_dim>>>(dst, r, V);
		// phase 2
		//Phase2<<<blocks, block_dim>>>(dst, r, V);
		Phase2_<<<grid_dim2, block_dim>>>(dst, r, V);
		// phase 3
		Phase3<<<grid_dim, block_dim>>>(dst, r, V);
		//Phase3_<<<grid_dim, block_dim>>>(dst, r, V);
	}

	cudaMalloc(&dst_, vertex*vertex*sizeof(int));
	for(int i = 0; i < vertex; ++i) {
		cudaMemcpy(dst_ + i*vertex, dst + i*V, sizeof(int)*vertex, cudaMemcpyDeviceToDevice);
    }
	cudaMemcpy(Dist, dst_, size_vertex, cudaMemcpyDeviceToHost);
	cudaFree(dst);
	cudaFree(dst_);
}
inline void input_(char* infile) {
	int file = open(infile, O_RDONLY);
	int *fpt = (int*)mmap(NULL, 2*sizeof(int), PROT_READ, MAP_PRIVATE, file, 0);
  	vertex = fpt[0];
	edge = fpt[1];
	int *pair = (int*)(mmap(NULL, (3 * edge + 2) * sizeof(int), PROT_READ, MAP_PRIVATE, file, 0));

	if (vertex % B){
        V = vertex + (B - vertex % B);//size of total matrix could be filled with B*B blocks
    } else {V = vertex;}
	Dist = (int*)malloc(V*V*sizeof(int));
	

	for (int i = 0; i < V; ++i) {
    	for (int j = 0; j < V; ++j) {
			Dist[i*V+j] = INF;
			if (i == j) Dist[i*V+j] = 0;
		}
    }

	for (int i = 0; i < edge; ++i) {
		Dist[pair[i*3+2]*V+pair[i*3+3]]= pair[i*3+4];//0,1 is vertex and edge
	}
	munmap(pair, (3 * edge + 2) * sizeof(int));//end the mapping
	close(file);
}
inline void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&vertex, sizeof(int), 1, file);
    fread(&edge, sizeof(int), 1, file);
    if (vertex % B){
        V = vertex + (B - vertex % B);//size of total matrix could be filled with B*B blocks
    } else {V = vertex;}
    Dist = (int*)malloc(V*V*sizeof(int));
    //fprintf(stderr, "edge:%d vertex:%d V:%d\n",edge,vertex,V);
    for (int i = 0; i < V; ++i) {
        for (int j = 0; j < V; ++j) {
                if (i == j){Dist[i*V+j] = 0;}
                else {Dist[i*V+j] = INF;}
                 
            }
    }
    

    int pair[3];
    for (int i = 0; i < edge; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*V+pair[1]] = pair[2];
        //printf("%d ",Dist[pair[0]*V+pair[1]]);
    }
    fclose(file);
}

inline void output_(char* outFileName) {
	//int *Dist_ = (int *)malloc(vertex * vertex * sizeof(int));

    FILE* outfile = fopen(outFileName, "w");
	
	fwrite(Dist, sizeof(int), vertex*vertex, outfile);
    fclose(outfile); 
}






int main(int argc, char* argv[]) {
    input_(argv[1]);
    //cudaGetDeviceProperties(&prop, DEV_NO);
    //printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    block_FW(V);
    output_(argv[2]);
    return 0;
}*/
