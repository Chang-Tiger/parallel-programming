
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8

/* Hint 7 */
// this variable is used by device
__constant__ int mask[MASK_N][MASK_X][MASK_Y] = { 
    {{ -1, -4, -6, -4, -1},
     { -2, -8,-12, -8, -2},
     {  0,  0,  0,  0,  0}, 
     {  2,  8, 12,  8,  2}, 
     {  1,  4,  6,  4,  1}},
    {{ -1, -2,  0,  2,  1}, 
     { -4, -8,  0,  8,  4}, 
     { -6,-12,  0, 12,  6}, 
     { -4, -8,  0,  8,  4}, 
     { -1, -2,  0,  2,  1}} 
};



int read_png(const char* filename, unsigned char** image, unsigned* height, 
             unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, 
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

/* Hint 5 */
// this function is called by host and executed by device
//void sobel (unsigned char* s, unsigned char* t, unsigned height, unsigned width, unsigned channels) {
__global__ void sobel(unsigned char *s, unsigned char *t, unsigned height, unsigned width, unsigned channels){

    int  x,  i, v, u;
    int  R, G, B;
    double val[MASK_N*3] = {0.0};
    
    int y = threadIdx.x + blockIdx.x * blockDim.x;//tid
    
    if(y > height) {
        return;
    }
    int adjustX, adjustY;
    int xBound = MASK_X /2;
    int yBound = MASK_Y /2;
    double totalR,totalG,totalB;
    // Hint 6 
    // parallel job by blockIdx, blockDim, threadIdx 
    //for (y = 0; y < height; ++y) {
    
    for (x = 0; x < width; ++x) {
        for (i = 0; i < MASK_N; ++i) {
            adjustX = (MASK_X % 2) ? 1 : 0;
            adjustY = (MASK_Y % 2) ? 1 : 0;
            

            val[i*3+2] = 0.0;
            val[i*3+1] = 0.0;
            val[i*3] = 0.0;

            for (v = -yBound; v < yBound + adjustY; ++v) {
                for (u = -xBound; u < xBound + adjustX; ++u) {
                    if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
                        R = s[channels * (width * (y+v) + (x+u)) + 2];
                        G = s[channels * (width * (y+v) + (x+u)) + 1];
                        B = s[channels * (width * (y+v) + (x+u)) + 0];
                        val[i*3+2] += R * mask[i][u + xBound][v + yBound];
                        val[i*3+1] += G * mask[i][u + xBound][v + yBound];
                        val[i*3+0] += B * mask[i][u + xBound][v + yBound];
                    }    
                }
            }
        }

        totalR = 0.0;
        totalG = 0.0;
        totalB = 0.0;
        for (i = 0; i < MASK_N; ++i) {
            totalR += val[i * 3 + 2] * val[i * 3 + 2];
            totalG += val[i * 3 + 1] * val[i * 3 + 1];
            totalB += val[i * 3 + 0] * val[i * 3 + 0];
        }

        totalR = sqrt(totalR) / SCALE;
        totalG = sqrt(totalG) / SCALE;
        totalB = sqrt(totalB) / SCALE;
        const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
        const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
        const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
        t[channels * (width * y + x) + 2] = cR;
        t[channels * (width * y + x) + 1] = cG;
        t[channels * (width * y + x) + 0] = cB;
    }
    
}

int main(int argc, char** argv) {

    assert(argc == 3);
    unsigned height, width, channels;
    unsigned char *src = NULL;
    unsigned char *dst;
    //unsigned char* host_s = NULL;
    read_png(argv[1], &src, &height, &width, &channels);
    dst = (unsigned char *)malloc(height * width * channels *sizeof(unsigned char));
    //unsigned char* host_t = (unsigned char*) malloc(height * width * channels * sizeof(unsigned char));
    
    // Hint 1 
    // cudaMalloc(...) for device src and device dst
    unsigned *d_height, *d_width, *d_channels;
    unsigned char *d_src, *d_dst;
    hipMalloc((void **)&d_height, sizeof(unsigned));
    hipMalloc((void **)&d_width, sizeof(unsigned));
    hipMalloc((void **)&d_channels, sizeof(unsigned));
    hipMalloc(&d_src, height * width * channels * sizeof(unsigned char));
    hipMalloc(&d_dst, height * width * channels * sizeof(unsigned char));
    // Hint 2 
    // cudaMemcpy(...) copy source image to device (filter matrix if necessary)
    hipMemcpy(d_height,&height,sizeof(unsigned),hipMemcpyHostToDevice);
    hipMemcpy(d_channels,&channels,sizeof(unsigned),hipMemcpyHostToDevice);
    hipMemcpy(d_width,&width,sizeof(unsigned),hipMemcpyHostToDevice);
    hipMemcpy(d_src, src, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Hint 3 
    // acclerate this function
    //sobel(host_s, host_t, height, width, channels);
    dim3 num_threads(64, 1); // Assuming 64 threads per block
    dim3 num_blocks(((width * height) / num_threads.x) + 1, 1);

    //dim3 num_blocks((width * height + num_threads.x - 1) / num_threads.x, 1);
    

    sobel<<<num_blocks, num_threads>>>(d_src, d_dst, height, width, channels);//<<<num_blocks, num_threads>>>
    // Hint 4
    // cudaMemcpy(...) copy result image to host
    hipMemcpy(dst, d_dst, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
    write_png(argv[2], dst, height, width, channels);
    hipFree(d_height);
    hipFree(d_width);
    hipFree(d_channels);
    hipFree(d_src);
    hipFree(d_dst);
    return 0;
}
